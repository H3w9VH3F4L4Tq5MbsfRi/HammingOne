#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <chrono>
#include <stdio.h>
#include <fstream>
#include <string>
#include <iostream>
#include <bitset>

#define size 64
#define threadsPerBlock 1024
#define megaToNormal 1000000

hipError_t calcWithCuda(const unsigned long long* data, const int n, const int rSize, const bool verbose);

__global__ void addKernel(const unsigned long long* data, const int n, const int rSize, const bool verbose, unsigned long long* pairs)
{
    int i = blockIdx.x * threadsPerBlock + threadIdx.x;
    if (i >= n)
        return;

    unsigned long long pairsCounter = 0;

    for (int j = 0; j < n; j++)
    {
        if (j == i)
            continue;

        bool singleDiffSpotted = false;
        bool multipleDiffSpotted = false;
        for (int k = 0; k < rSize; k++)
        {
            if (multipleDiffSpotted)
                break;
            unsigned long long xored = data[i + n * k] ^ data[j + n * k];
            if (xored == 0)
                continue;
            unsigned long long testValue = xored & (xored - 1);
            if (testValue == 0 && !singleDiffSpotted)
            {
                singleDiffSpotted = true;
                continue;
            }
            multipleDiffSpotted = true;
        }

        if (!multipleDiffSpotted && singleDiffSpotted)
        {
            if (verbose)
                printf("%d is a pair with %d \n", i, j);
            pairsCounter++;
        }
    }
    pairs[i] = pairsCounter;

    return;
}

int main(int argc, char** argv)
{
    std::cout << "Patryk Saj" << std::endl;
    std::cout << "GPU Project 1" << std::endl;
    std::cout << "Hamming one" << std::endl << std::endl;

    if (argc < 2 || argc > 4)
    {
        std::cout << "Invalid parameters!" << std::endl;
        std::cout << "Terminating program..." << std::endl;
        return 1;
    }

    //Testing provided file for expected dataformat
    std::string path = argv[1];
    int n;
    int l;
    try
    {
        std::ifstream file(path);
        std::string str;
        std::getline(file, str);
        std::string nStr = str.substr(0, str.find(','));
        n = stoi(nStr);
        std::string lStr = str.substr(str.find(',') + 1, str.length());
        l = stoi(lStr);
    }
    catch (...)
    {
        std::cout << "Unable to read provided file!" << std::endl;
        std::cout << "Terminating program..." << std::endl;
        return 1;
    }

    // Reading data
    std::cout << "Reading from " << path << std::endl;
    std::cout << "n = " << n << std::endl << "l = " << l << std::endl;

    int rSize = (int)ceil((double)l / size);
    unsigned long long* data = new unsigned long long[rSize * n];
    std::ifstream file(path);
    std::string str;
    std::string str2;
    std::getline(file, str);

    auto start0 = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < n; i++)
    {
        std::getline(file, str);
        for (int j = 0; j < rSize; j++)
        {
            str2 = str.substr(j * size, size);
            data[i + n * j] = std::bitset<size>(str2).to_ullong();
        }
    }
    auto stop0 = std::chrono::high_resolution_clock::now();
    auto duration0 = std::chrono::duration_cast<std::chrono::microseconds>(stop0 - start0);
    std::cout << "Data loading took " << duration0.count() / (double)megaToNormal << " s." << std::endl;

    //Getting parameters
    bool CPUversion = false;
    bool verbose = false;
    for (int i = 2; i < 4; i++)
    {
        if (argc > i)
        {
            std::string param = argv[i];

            if (param[0] == '-')
            {
                if (param[1] == 'v' && verbose == false)
                    verbose = true;
                else if (param[1] == 'c' && CPUversion == false)
                    CPUversion = true;
                else
                {
                    std::cout << "Invalid parameters!" << std::endl;
                    std::cout << "Terminating program..." << std::endl;
                    return 1;
                }
            }
            else
            {
                std::cout << "Invalid parameters!" << std::endl;
                std::cout << "Terminating program..." << std::endl;
                return 1;
            }
        }
    }

    if (CPUversion)
    {
        // CPU version of the algorithm
        std::cout << std::endl << "CPU version of the algorythm:" << std::endl;

        unsigned long long pairsCounter = 0;
        auto start = std::chrono::high_resolution_clock::now();
        for (int i = 0; i < n; i++)
        {
            for (int j = 0; j < n; j++)
            {
                if (j == i)
                    continue;

                bool singleDiffSpotted = false;
                bool multipleDiffSpotted = false;
                for (int k = 0; k < rSize; k++)
                {
                    if (multipleDiffSpotted)
                        break;
                    unsigned long long xored = data[i + n * k] ^ data[j + n * k];
                    if (xored == 0)
                        continue;
                    unsigned long long testValue = xored & (xored - 1);
                    if (testValue == 0 && !singleDiffSpotted)
                    {
                        singleDiffSpotted = true;
                        continue;
                    }
                    multipleDiffSpotted = true;
                }

                if (!multipleDiffSpotted && singleDiffSpotted)
                {
                    if (verbose)
                        std::cout << i << " is a pair with " << j << std::endl;
                    pairsCounter++;
                }
            }
        }

        auto stop = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
        std::cout << "CPU version execution time: " << duration.count() / (double)megaToNormal << " s" << std::endl;
        std::cout << "Found " << pairsCounter / 2 << " pairs." << std::endl;
    }

    //GPU version of the algorithm
    std::cout << std::endl << "GPU version of the algorythm:" << std::endl;
    hipError_t cudaStatus = calcWithCuda(data, n, rSize, verbose);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

hipError_t calcWithCuda(const unsigned long long* data, const int n, const int rSize, const bool verbose)
{
    unsigned long long* dev_data = 0;
    unsigned long long* dev_pairs = 0;
    hipError_t cudaStatus;
    int count = (int)ceil((double)n / threadsPerBlock);

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_data, n * rSize * sizeof(unsigned long long));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_pairs, n * sizeof(unsigned long long));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_data, data, n * rSize * sizeof(unsigned long long), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    auto start2 = std::chrono::high_resolution_clock::now();
    addKernel <<< count, threadsPerBlock >>> (dev_data, n, rSize, verbose, dev_pairs);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    auto stop2 = std::chrono::high_resolution_clock::now();
    auto duration2 = std::chrono::duration_cast<std::chrono::microseconds>(stop2 - start2);
    std::cout << "GPU version execution time: " << duration2.count() / (double)megaToNormal << " s" << std::endl;

    unsigned long long* GPUpairs = new unsigned long long[n];
    cudaStatus = hipMemcpy(GPUpairs, dev_pairs, n * sizeof(unsigned long long), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    unsigned long long counter = 0;
    for (int i = 0; i < n; i++)
        counter += GPUpairs[i];
    std::cout << "Found " << counter / 2 << " pairs." << std::endl;


Error:
    hipFree(dev_data);
    hipFree(dev_pairs);

    return cudaStatus;
}
